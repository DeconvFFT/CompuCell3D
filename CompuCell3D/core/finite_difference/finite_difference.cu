#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>
#include<finite_difference.h>
#include<vector>

using namespace std;
// stencil coefficients
__constant__ float c_ax, c_ay; // constant multiplier
dim3 grid[3][2], block[3][2]; // initialise grid and block config

// function for checking CUDA runtime API
inline
HRESULT cudaSafeCall(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    printf(E_FAIL, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    //assert(result == hipSuccess);
    return E_FAIL;
  }
#endif
  return S_OK;
}


// set parameter values
// returns 1 on success, 
HRESULT setDerivativeParameters(int mx, int my, int mz, int sPencils, int lPencils){

  if((mx %sPencils!=0) || (my %sPencils!=0) || (mz %sPencils!=0)){
    
    // replace with error macro
    printf("'mx', 'my', and 'mz' must be integral multiples of sPencils\n", "got error");
    return E_FAIL; 
  }

  if ((mx % lPencils != 0) || (my % lPencils != 0)) {
    printf("'mx' and 'my' must be multiples of lPencils\n", "got error");
    return E_FAIL; 
  }

  float dx = mx-1; // asigning a value for deltaX
  float k = 1.f/5.f; // assign a value to constant k
  float dt = 2; // delta t is assumed to be 2 units 
  float ax = (k * dt) / (dx * dx);

  cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ax), &ax, sizeof(float), 0, hipMemcpyHostToDevice));

  float dy = my-1; 
  float ay = (k * dt) / (dy * dy);

  cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ay), &ay, sizeof(float), 0, hipMemcpyHostToDevice));

  // grid and block execution configurations


  grid[0][0] = dim3(my/sPencils, mz, 1);

  block[0][0] = dim3(mx, sPencils, 1);

  grid[0][1] = dim3(my/lPencils, mz, 1);

  block[0][1] = dim3(mx, sPencils, 1);

  grid[1][0] = dim3(mx/sPencils, mz, 1);

  block[1][0] = dim3(sPencils, my, 1);


  grid[1][1] = dim3(mx/lPencils, mz, 1);

  block[1][1] = dim3(lPencils, my*sPencils/lPencils, 1);

  grid[2][0] = dim3(mx/sPencils, my, 1);

  block[2][0] = dim3(sPencils, mz, 1);

  grid[2][1] = dim3(mx/lPencils, my, 1);

  block[2][1] = dim3(lPencils, mz*sPencils/lPencils, 1);

return S_OK;
}

// inputs:
// pointer to float vector, pointer to derivative vector
__global__ void derivativeX(float *f, float *df, int mx, int my){
  int neighbours=2;
  //  int dim2 = (int)(sf_dim/sPencils);
  // __shared__ float s_f[sPencils][dim2]; // handles upto 2 neighbours for now
  // one dim => float ptrs to first elementg of arrays
 extern __shared__ float s_f []; // dynamic shared memory.
  //float * p1 = &s_f[0];
  int offset = (int)(neighbours/2); // offset for memory access

  // thread and block indexes
  int i = threadIdx.x;
  int j = blockIdx.x*blockDim.y +threadIdx.y;
  int si = i+offset; // local i for shared memory access + halo offset
  int k  = blockIdx.y;

  // int sj = threadIdx.y; // local j for shared memory access
  int index = si+(threadIdx.y*mx);
  int globalIdx = k*mx*my + j*mx+i;
   // s_f[sj][si] = f[globalIdx];
  s_f[index] = f[globalIdx];
  __syncthreads();


  // get derivative
 //df[globalIdx] = c_ax *(s_f[sj][si-1] - 2*s_f[sj][si] + s_f[sj][si+1]) // single dim araays here too
  df[globalIdx] = c_ax *(s_f[index-mx] - 2*s_f[index] + s_f[index+mx]); // single dim araays here too

}

// derivative y

__global__ derivativeY(float *f, float *df, int mx, int my){
   int neighbours=2;
  //  int dim2 = (int)(sf_dim/sPencils);
  // __shared__ float s_f[sPencils][dim2]; // handles upto 2 neighbours for now
  // one dim => float ptrs to first elementg of arrays
  extern __shared__ float s_f []; // dynamic shared memory.
  //float * p1 = &s_f[0];
  int offset = (int)(neighbours/2); // offset for memory access


  // thread and block indexes
  int i = blockIdx.x*blockDim.x +threadIdx.x;
  int j = threadIdx.y;
  
  int si = i+offset; // local i for shared memory access + halo offset
  int sj = threadIdx.y; // local j for shared memory access

  // int globalIdx = k*mx*my + j*mx+i;
  // int index = 
  // s_f[sj][si] = f[globalIdx];


 int index = si+(threadIdx.y*mx);
 int globalIdx = k*mx*my + j*mx+i;
 s_f[index] = f[globalIdx];
  __syncthreads();


  // get derivative
  // df[globalIdx] = c_ax *(s_f[sj][si-1] - 2*s_f[sj][si] + s_f[sj][si+1]) 
  df[globalIdx] = c_ay *(s_f[index-mx] - 2*s_f[index] + s_f[index+mx]); // single dim araays here too

}
// @param: data, takes a vector of floats as input
// can be a derivative across any dimension
// assunme 3d array structure for now
// input to the funciton: only rhew aRRAY PART (floatfieldImpl -> cc3d.cpp.compucell. field from Field3D, a 1D array)

// ** april 14, 2023 notes: **
// overload which looks for a double vector 
// divide the function into blocking and non blocking calls
 

HRESULT derivativeDevice(std::vector<float> &data){
float *f = &data[0];
float *d_f,*d_df;
const int mx=64, my=64, mz=64; // block size on one axis
float *df = new float[mx*my*mz];

// shared memory tiles will have: m* x *pencil
// spencil is used for derivate at one point in one direction
// lpencil is used for coalescing in y and z direction 
// where each thread calculkates derivate at multiple points
const int sPencils = 1; // 1 neighbour each side
const int lPencils = 32;
int sfSizeX = sPencils * (mx+2) * sizeof(float); 


int n_bytes =  sizeof(float)* mx*my*mz;

cudaSafeCall(hipMalloc((float **)&d_f, n_bytes));

cudaSafeCall(hipMalloc((float **)&d_df, n_bytes));


setDerivativeParameters(mx, my, mz, sPencils, lPencils); // set parameters for device

// switch case for which dimension:

// calculations for X derivative
int dimension = 0; 

// copy float to device and initialise derivative of floats

cudaSafeCall(hipMemcpy(d_f, f, n_bytes, hipMemcpyHostToDevice));  

cudaSafeCall(hipMemset(d_df, 0, n_bytes));


derivativeX<<<grid[dimension][0],block[dimension][0], sfSizeX>>>(d_f, d_df, mx,my); // get x derivative

cudaSafeCall( hipMemcpy(df, d_df, n_bytes, hipMemcpyDeviceToHost) );

cudaSafeCall(hipFree(d_f)); // free floats on device

cudaSafeCall(hipFree(d_df)); // free derivatives on device

// delete [] f;

return S_OK;
}

HRESULT derivativeDevice(std::vector<double> &data){
  
    std::vector<float> temp(data.begin(), data.end());
     if (temp.size() != data.size()) {
      return E_FAIL;
     }
    return derivativeDevice(temp);
}

// kernel for SAXPY
__global__
void simpleAdd(int n, float a, float *x, float *y){
    int tId = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=tId; i<n; i+=stride){
        y[i] = a * x[i] + y[i];
    }
}


HRESULT simpleAddDevice(){
  // allocate data on device
  int size = 1<<20;
  int blockSize = 256;
  int n_bytes = sizeof(float) * size;
  float *h_x, *h_y;
  float a = 2.0f;

  h_x = (float *) malloc(n_bytes);
  h_y = (float *) malloc(n_bytes);
  // host memory initialisation
  for(int i=0;i<size;i++){
    h_x[i] = 10.0f;
    h_y[i] = 20.0f;
  }

  // device memory initialisation
  float *d_x, *d_y;
  cudaSafeCall(hipMalloc((float **)&d_x, n_bytes));
  cudaSafeCall(hipMalloc((float **)&d_y, n_bytes));
  
  // copy host data necessary to device
  cudaSafeCall(hipMemcpy(d_x,h_x, n_bytes, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(d_y,h_y, n_bytes, hipMemcpyHostToDevice));
  
  // call kernbel with device side ptrs
  dim3 block(blockSize);
  dim3 grid(size/block.x);
  //<<< grid, block , amt_shared_memory,,stream(schedule calulations)>>>
  // mulkti threading like setup
  simpleAdd<<< grid, block >>>(size, a, d_x, d_y);
  cudaSafeCall(hipDeviceSynchronize());
  
  // copy back results to host
  cudaSafeCall(hipMemcpy(d_y, h_y, n_bytes, hipMemcpyDeviceToHost));
  
  // freeup memory allocated
  free(h_x);
  free(h_y);
  
  cudaSafeCall(hipFree(d_x));
  cudaSafeCall(hipFree(d_y));
  return S_OK;
}