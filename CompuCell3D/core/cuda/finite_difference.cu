#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>
#include<finite_difference.h>
// function for checking CUDA runtime API
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__
void simpleAdd(int n, float a, float *x, float *y){
    int tId = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=tId; i<n; i+=stride){
        y[i] = a * x[i] + y[i];
    }
}

void simpleAddDevice(){
  // allocate data on device
  int size = 1<<20;
  int blockSize = 256;
  int n_bytes = sizeof(float) * size;
  float *h_x, *h_y;
  float a = 2.0f;

  h_x = (float *) malloc(n_bytes);
  h_y = (float *) malloc(n_bytes);
  // host memory initialisation
  for(int i=0;i<size;i++){
    h_x[i] = 10.0f;
    h_y[i] = 20.0f;
  }

  // device memory initialisation
  float *d_x, *d_y;
  hipMalloc((float **)&d_x, n_bytes);
  hipMalloc((float **)&d_y, n_bytes);
  
  // copy host data necessary to device
  hipMemcpy(d_x,h_x, n_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y,h_y, n_bytes, hipMemcpyHostToDevice);
  
  // call kernbel with device side ptrs
  dim3 block(blockSize);
  dim3 grid(size/block.x);
  simpleAdd<<< grid, block >>>(size, a, d_x, d_y);
  hipDeviceSynchronize();
  
  // copy back results to host
  hipMemcpy(d_y, h_y, n_bytes, hipMemcpyDeviceToHost);
  
  // freeup memory allocated
  free(h_x);
  free(h_y);
  
  hipFree(d_x);
  hipFree(d_y);
}